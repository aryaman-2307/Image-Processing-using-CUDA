#include "img_utils.hpp"

#include <hip/hip_runtime.h>
#include <nppi_filtering_functions.h>
#include <sys/stat.h>

#define STB_IMAGE_IMPLEMENTATION
#include "../third_party/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../third_party/stb_image_write.h"

#include <algorithm>
#include <filesystem>
#include <iostream>

namespace fs = std::filesystem;
namespace img {


auto ReadGray(const std::string& path, int* w, int* h) -> std::vector<uint8_t> {
    int channels;
    uint8_t* data = stbi_load(path.c_str(), w, h, &channels, 1);
    if (!data) { throw std::runtime_error("Failed to read " + path); }
    std::vector<uint8_t> v(data, data + (*w) * (*h));
    stbi_image_free(data);
    return v;
}

void WriteGray(const std::string& path,
               int w,
               int h,
               const uint8_t* data) {
    if (!stbi_write_png(path.c_str(), w, h, 1, data, w)) {
        throw std::runtime_error("Failed to write " + path);
    }
}


void MedianFilter(const uint8_t* src,
                  uint8_t*       dst,
                  int            width,
                  int            height,
                  int            radius) {
    size_t pitch;
    uint8_t* d_src;
    uint8_t* d_dst;
    hipMallocPitch(&d_src, &pitch, width, height);
    hipMallocPitch(&d_dst, &pitch, width, height);
    hipMemcpy2D(d_src, pitch, src, width, width, height, hipMemcpyHostToDevice);

    NppiSize oSize{width, height};
    NppiSize oMask{2 * radius + 1, 2 * radius + 1};
    NppiPoint oAnchor{radius, radius};

    NppiSize oRoi{width - 2 * radius, height - 2 * radius};

    Npp8u* pTmp;
    size_t tmp_bytes;
    nppiFilterMedianGetBufferHostSize_8u_C1R(oRoi, oMask, &tmp_bytes);
    hipMalloc(&pTmp, tmp_bytes);

    NppStatus st = nppiFilterMedian_8u_C1R(
        d_src + pitch * radius + radius, pitch,
        d_dst + pitch * radius + radius, pitch,
        oRoi, oMask, oAnchor, pTmp);
    if (st != NPP_SUCCESS) throw std::runtime_error("NPP failure");

    hipMemcpy2D(dst, width, d_dst, pitch, width, height, hipMemcpyDeviceToHost);
    hipFree(pTmp);
    hipFree(d_src);
    hipFree(d_dst);
}

std::vector<std::string> ListImageFiles(const std::string& dir_path) {
    std::vector<std::string> files;
    for (const auto& e : fs::directory_iterator(dir_path)) {
        if (!e.is_regular_file()) continue;
        std::string ext = e.path().extension().string();
        std::transform(ext.begin(), ext.end(), ext.begin(), ::tolower);
        if (ext == ".png" || ext == ".jpg" || ext == ".jpeg") {
            files.push_back(e.path().string());
        }
    }
    std::sort(files.begin(), files.end());
    return files;
}

} 
